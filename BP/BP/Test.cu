#include <hip/hip_runtime.h>


__global__ void MatrixMulKernel(float * Md, float * Nd, float * Pd, int Width)
{
    // identifiant de thread � deux dimensions, comme la matrice
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    // Pvaleur sert au stockage de la valeur calcul�e par le thread
    float Pvaleur = 0;
    for (int i = 0; i < Width; ++i)
    {
        float MdElement = Md[ty * Width + i];
        float NdElement = Nd[i  * Width + tx];
        Pvaleur        += MdElement * NdElement;
    }
    // �crit la valeur calcul�e dans la matrice de r�sultat
    // chaque thread ne peut �crire qu'une valeur !
    Pd[ty * Width + tx] = Pvaleur;
}

void MatrixMulOnDevice(float * M, float * N, float * P, int Width)
{
    //calcul de la taille des matrices
    int size = Width * Width * sizeof(float);

	float *Md;
	float *Nd;
	float *Pd;

    //allocation des matrices et leur remplissage
    hipMalloc((void**) &Md, size);
    hipMemcpy(Md, M, size, hipMemcpyHostToDevice) ;
    hipMalloc((void**) &Nd, size);
    hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);

    //allocation de la matrice de r�sultat
    hipMalloc((void**) &Pd, size);

    //multiplication d'une seule matrice
    dim3 dimGrid(1, 1);
    //matrice carr�e
    dim3 dimBlock(Width, Width);

    //produit matriciel proprement dit
    MatrixMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd, Width);

    //r�cup�ration du r�sultat du calcul
    hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);

    //destruction des matrices, d�sormais inutilis�es
    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);
}